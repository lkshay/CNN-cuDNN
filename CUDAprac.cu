#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <cstdlib>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <string>
#include <vector>
using namespace std;
using namespace cv;

#define N 102400

// --- Define a function for cuDNN function status returns --- //

#define checkCUDNN(expression)                               \
  {                                                          \
    hipdnnStatus_t status = (expression);                     \
    if (status != HIPDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << hipdnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }

// --- Define a function for cuDNN function error returns --- //

#define checkCudaErrors(status) do {                         \
    std::stringstream _error;                                \
    if (status != 0) {                                       \
      _error << "Cuda failure: " << status;                  \
      FatalError(_error.str());                              \
    }                                                        \
} while(0)


__global__ void kernel1(int *a, int num){

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(int i = idx; i < num; i += stride){

		a[i] = 2;
		//__syncthreads();
	}

}


int device = 0;

int main(int argc, const char* argv[]){

	int *a;
	
	hipMallocManaged(&a, N*sizeof(int));

	hipSetDevice(device);

	hipGetDevice(&device);
	cout<<device<<endl;

	hipMemPrefetchAsync(a, N*sizeof(int), device);

	int threads_per_block = 1024;
	int numBlocks = (N + threads_per_block - 1)/threads_per_block;

	kernel1<<<threads_per_block,numBlocks>>>(a,(int)N);

	hipMemPrefetchAsync(a,N*sizeof(int),hipCpuDeviceId);
	
	hipDeviceSynchronize();

	int error = 0;
	for(int i = 0; i < N; i++){

		if(a[i] != 2){
			error = 1;
		}
		//cout<<a[i] << " "<< i<< endl;
	}

	cout << error << endl;
	hipFree(a);
	return 0;
}