#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <string>
#include <random>
#include <cmath>
#include <stdio.h>
using namespace std;
using namespace cv;


#define BATCH_SIZE 1
#define OVERLAP_POOLING 1
#define BIAS_INIT_VAL 0.001
#define MAX_THREADS_PER_BLOCK 1024 // according to GTX 1050 Ti

int gpu_id;
int device;

int roundUp(int num, int den){

  return((num + den - 1 )/(den));

}

// a struct for outputs & inputs of convolutional layers (including max pool, if exists)

struct convDim_t{

  int Height;
  int Width;
  int Channels;
  int Batch;
};

// a struct to define pooling layers for a conv layer (if pool is true)
struct poolDim_t{

  int Height;
  int Width;
  int padHeight;
  int padWidth;
  int strideHeight;
  int strideWidth;
};

// a struct to define kernel dimensions for a conv operation in a layer

struct kernelDim_t{

  int kernelSize;
  int kernelHeight;
  int kernelWidth;
  int strideHeight;
  int strideWidth;
  int padHeight;
  int padWidth;
  int dilationHeight;
  int dilationWidth;
};


// a function to set the kernel params for a conv operation in a lyer

kernelDim_t setKernelSpecs(int size, int fheight, int fwidth, int sheight, int swidth, int pheight, int pwidth, int dheight, int dwidth){

  kernelDim_t layerKernel;
  layerKernel.kernelSize = size;
  layerKernel.kernelHeight = fheight;
  layerKernel.kernelWidth = fwidth;
  layerKernel.strideHeight = sheight;
  layerKernel.strideWidth = swidth;
  layerKernel.padHeight = pheight;
  layerKernel.padWidth = pwidth;
  layerKernel.dilationHeight = dheight;
  layerKernel.dilationWidth = dwidth;

  return layerKernel;
}

/*
int flagOverlap is a flag for setting dimensions ov poolDims. If it is 1, then F=3,S=2 otherwise F=2,S=2.
It is worth noting that there are only two commonly seen variations of the max pooling layer found in practice: 
A pooling layer with F=3,S=2 (also called overlapping pooling), and more commonly F=2,S=2. Pooling sizes with larger receptive fields are too destructive.
*/

// a function to set pool dimensions for a layer operation, if pool is true

poolDim_t setPoolSpecs(bool flagOverlap){

  poolDim_t poolDims;

  if(flagOverlap){

    poolDims.Height = 3;
    poolDims.Width = 3;
    poolDims.padHeight = 1;
    poolDims.padWidth = 1;
    poolDims.strideHeight = 2;
    poolDims.strideWidth = 2;  
  }
  else{
    poolDims.Height = 2;
    poolDims.Width = 2;
    poolDims.padHeight = 1;
    poolDims.padWidth = 1;
    poolDims.strideHeight = 2;
    poolDims.strideWidth = 2;
  }

  return poolDims;
  
}


#define checkCUDNN(expression)                             \
{                                                          \
  hipdnnStatus_t status = (expression);                     \
  if (status != HIPDNN_STATUS_SUCCESS) {                    \
    std::cerr << "Error on line " << __LINE__ << ": "      \
              << hipdnnGetErrorString(status) << std::endl; \
    std::exit(EXIT_FAILURE);                               \
  }                                                        \
}


// --- A function to convert the image to a array to be passed into the input conv layer --- //

float * image2array(Mat image){

  float *imageArray = (float *)image.data;
  
  return imageArray;
}

void save_image(const string output_filename,
                float* buffer,
                int height,
                int width) {
  cv::Mat output_image(height, width, CV_32FC3, buffer);
  // Make negative values zero.
  cv::threshold(output_image,
                output_image,
                /*threshold=*/0,
                /*maxval=*/0,
                cv::THRESH_TOZERO);
  cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
  output_image.convertTo(output_image, CV_8UC3);
  cv::imwrite(output_filename, output_image);
  std::cerr << "Wrote output to " << output_filename << std::endl;
}

Mat load_image(const char* image_path) {
  cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_COLOR);
  image.convertTo(image, CV_32FC3);
  cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
  cerr << "Input Image: " << image.rows << " x " << image.cols << " x "
          << image.channels() << endl;
  return image;
}

/*

-Prepare the kernel and bias tensors before sending them to the next layer. Initialize the kernel with He. et. al., and bias with zero.
-The filter initializer is common to all the layers, so is the bias. The channel layout is also same except in some cases (will find out later)
-Input to the layer is now the dimension of kernel tensor, bias tensor, input tensor, and the initialized bias and kernel tensors and the
 and the input tensor either from a previous layer or the input image

*/

float alpha = 1.0;
float beta = 0.0;

class ConvLayers{

	// In addition to the inHeight, inWidth and inChannels pertaining to the output of the previous layer,
	// also have output object of the previous layer as a member of this class.
  public:

  float *inputTensor;                   // pointer to the input tensor. If this is an input layer, convert the cv::Mat image to a 3-D array first and then pass its pointer 
                                        // to the class constructor 
  float *kernelTensor;
  float *biasTensor;		
  int layerIndex;
  float alph, bet;
  hipdnnHandle_t CUDNN;
  hipdnnTensorFormat_t TensorFormat;
  hipdnnDataType_t DataType;
  hipdnnConvolutionMode_t ConvMode;
  hipdnnActivationMode_t ActivationMode;
  hipdnnPoolingMode_t PoolingMode;
  convDim_t outDims;
  convDim_t inDims;
  kernelDim_t kernelDims;
  poolDim_t poolDims;

  random_device rd{};
  mt19937 gen{rd()};  
  normal_distribution<> d{0,1}; 

  float* conv_output{nullptr}; // output of convolution operation
  float* poolTensor{nullptr};  // output of pooling layer, if exists
  float* outputTensor{nullptr};
  void* d_workspace{nullptr};
  size_t workspaceBytes{0};

  int convOutDimHeight{0}, convOutDimWidth{0}, convOutDimChannels{0}, convOutDimBatchSize{0};
  int poolOutBatchSize{0}, poolOutChannels{0}, poolOutHeight{0}, poolOutWidth{0};

  bool POOL;  // True if pooling is to be done in this layer, otherwise False

  hipdnnTensorDescriptor_t input_descriptor;
  hipdnnFilterDescriptor_t kernel_descriptor;
  hipdnnConvolutionDescriptor_t convolution_descriptor;
  hipdnnTensorDescriptor_t bias_descriptor;
  hipdnnTensorDescriptor_t convOutput_descriptor;
  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  hipdnnActivationDescriptor_t activation_descriptor;
  hipdnnPoolingDescriptor_t pooling_descriptor;
  hipdnnTensorDescriptor_t poolTensor_descriptor;

	  
  /*
  Constructor overloading for initialiing the class objefloat *kernelTensor;
  float *biasTensor;ct with or without pooling mode. If the user wants to use pooling layer, use the second signature, otherwise first.
  The ConvLayers class' objects behave differently when a pool layer is to be used and differently when pool isnt there!
  */

  	// Empty constructor for subclass FCLayers
  ConvLayers(){}


	ConvLayers( int index, float* inT, convDim_t inDim, kernelDim_t kdims, float a, float b, 
		hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type, hipdnnConvolutionMode_t c_mode, hipdnnActivationMode_t ActMode, hipdnnHandle_t cud){

    this->POOL = false;
    this->inputTensor = inT;
    this->inDims = inDim;
    this->kernelDims = kdims;

    this->layerIndex = index;
    this->alph = a; this->bet = b;
    this->TensorFormat = t_format;
    this->DataType = d_type;			
    this->ConvMode = c_mode;
    this->ActivationMode = ActMode;
    this->CUDNN = cud;	
	}

  ConvLayers( int index, float* inT, convDim_t inDim, kernelDim_t kdims, poolDim_t pdims, float a, float b, 
    hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type, hipdnnConvolutionMode_t c_mode, hipdnnActivationMode_t ActMode,hipdnnPoolingMode_t poolMode, hipdnnHandle_t cud){

    this->POOL = true;
    this->inputTensor = inT;
    this->inDims = inDim;
    this->kernelDims = kdims;
    this->poolDims = pdims;
    this->layerIndex = index;
    this->alph = a; this->bet = b;
    this->TensorFormat = t_format;
    this->DataType = d_type;      
    this->ConvMode = c_mode;
    this->ActivationMode = ActMode;
    this->PoolingMode = poolMode;
    this->CUDNN = cud;  
  }

  	void getConvLayerSpecs();

	void buildConvLayer();

  	void fwdProp();

  	void bwdProp();

};

  void ConvLayers::getConvLayerSpecs(){



  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                          TensorFormat,
                                          DataType,
                                          inDims.Batch,
                                          inDims.Channels, 
                                          inDims.Height, 
                                          inDims.Width));

   
  // --- Build the Kernel which is going to convolve over the input ---//
  
  
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        DataType,
                                        TensorFormat,
                                        kernelDims.kernelSize,
                                        inDims.Channels,
                                        kernelDims.kernelHeight,
                                        kernelDims.kernelWidth));

  // --- Build the Convolution descriptor --- //

  
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                            kernelDims.padHeight,
                                            kernelDims.padWidth,
                                            kernelDims.strideHeight,
                                            kernelDims.strideWidth,
                                            kernelDims.dilationHeight,
                                            kernelDims.dilationWidth,
                                            ConvMode,
                                            DataType));

  // --- This function returns the dimensions of the resulting 4D tensor of a 2D convolution,     //
  // ---given the convolution descriptor, the input tensor descriptor and the filter descriptor --- //

  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                 input_descriptor,
                                                 kernel_descriptor,
                                                 &convOutDimBatchSize,
                                                 &convOutDimChannels,
                                                 &convOutDimHeight,
                                                 &convOutDimWidth));
  
  outDims.Height = convOutDimHeight;
  outDims.Width = convOutDimWidth;
  outDims.Channels = convOutDimChannels;
  outDims.Batch = convOutDimBatchSize;
  
  cout<<"Output image size "<<outDims.Batch<<" X "<<outDims.Height<<" X "<<outDims.Width<<" X "<<outDims.Channels<<endl;
  
  checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(bias_descriptor,
                                            TensorFormat,
                                            DataType,
                                            convOutDimBatchSize,
                                           convOutDimChannels,
                                           convOutDimHeight,
                                           convOutDimWidth));

  // ---Build the output Descriptor ---//

  
  checkCUDNN(hipdnnCreateTensorDescriptor(&convOutput_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(convOutput_descriptor,
                                        TensorFormat,
                                        DataType, 
                                        convOutDimBatchSize,
                                        convOutDimChannels,
                                        convOutDimHeight,
                                        convOutDimWidth));

  // -- Size references for next conv layer --- //

  

  // --- Determine the Convolution algorithm to be used in CNN layer ---//

  
  checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CUDNN,
                                        input_descriptor,
                                        kernel_descriptor,
                                        convolution_descriptor,
                                        convOutput_descriptor,
                                        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                        /*memoryLimitInBytes=*/0,
                                        &convolution_algorithm));

  
  checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
  checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
                                        ActivationMode,
                                        HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/0));


  /*
  Do some adjustment if the output dimension of pooling layer is not an integer (which will give an error) 
  Each dimension h and w of the output images is computed as followed:
  outputDim = 1 + (inputDim + 2*padding - windowDim)/poolingStride;

  */

  // check if the user has asked to create a pooling layer for this conv layer
  if(POOL){

    if((outDims.Height - poolDims.Height)%2 != 0){
      poolDims.Height = (poolDims.Height == 2) ? 3 : 2;
    }

    if((outDims.Width - poolDims.Width)%2 != 0){
      poolDims.Width = (poolDims.Width == 2) ? 3 : 2;
    }

    
    checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_descriptor));
    checkCUDNN(hipdnnSetPooling2dDescriptor(pooling_descriptor,
                                            PoolingMode,
                                            HIPDNN_NOT_PROPAGATE_NAN,
                                            poolDims.Height,
                                            poolDims.Width,
                                            poolDims.padHeight,
                                            poolDims.padWidth,
                                            poolDims.strideHeight,
                                            poolDims.strideWidth));

    checkCUDNN(hipdnnGetPooling2dForwardOutputDim(pooling_descriptor,
                                              convOutput_descriptor,
                                                  &poolOutBatchSize,
                                                  &poolOutChannels,
                                                  &poolOutHeight,
                                                  &poolOutWidth));

    
    checkCUDNN(hipdnnCreateTensorDescriptor(&poolTensor_descriptor));  
    checkCUDNN(hipdnnSetTensor4dDescriptor(poolTensor_descriptor,
                                          TensorFormat,
                                          DataType,
                                          poolOutBatchSize,
                                          poolOutChannels,
                                          poolOutHeight,
                                          poolOutWidth));

    outDims.Batch = poolOutBatchSize;
    outDims.Channels = poolOutChannels;
    outDims.Height = poolOutHeight;
    outDims.Width = poolOutWidth;

    }
  }

  void ConvLayers::buildConvLayer(){

  	// --- Set up the memory required for the convolution --- //
  
	  
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CUDNN,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     convOutput_descriptor,
                                                     convolution_algorithm,
                                                     &workspaceBytes));

    // Initialize bias and kernel tensors here //

    // Bias
    hipMallocManaged(&biasTensor, outDims.Channels * outDims.Batch * sizeof(float));
    hipMemset(biasTensor,(float)BIAS_INIT_VAL,outDims.Channels * outDims.Batch * sizeof(float)); //initializing all the bias units to BIAS_INIT_VAL

    //Kernel
    //random_device rd{};
    //mt19937 gen{rd()};

    // for initialization of weights with gassian distribution with zero mean and variance as one
    //normal_distribution<> d{0,1}; 

    // callibrator to be multplied with the weights for scaling according to He. et. al. 
    float callibrator = (layerIndex != 1) ? sqrt(2 / (inDims.Channels * kernelDims.kernelHeight * kernelDims.kernelWidth)) : 1.0;

    float kernelTemplate[kernelDims.kernelHeight][kernelDims.kernelWidth];
    for(int i = 0; i < kernelDims.kernelHeight; i++){
      for(int j = 0; j < kernelDims.kernelWidth; j++){
        kernelTemplate[i][j] = d(gen) * callibrator;
      }
    }
    float hkernel[kernelDims.kernelSize][inDims.Channels][kernelDims.kernelHeight][kernelDims.kernelWidth];

    for(int i = 0; i < kernelDims.kernelSize; i++){
      for(int j = 0; j < inDims.Channels; j++){
        for(int k = 0; k < kernelDims.kernelHeight; k++){
          for(int l = 0; l < kernelDims.kernelWidth; l++){
            hkernel[i][j][k][l] = kernelTemplate[k][l]; 
          }
        }
      }
    }

    hipMallocManaged(&kernelTensor, kernelDims.kernelSize * kernelDims.kernelHeight * kernelDims.kernelWidth * sizeof(float));
    hipMemcpy(kernelTensor,hkernel,sizeof(hkernel),hipMemcpyHostToDevice);
    
    // --- Allocate Memory in the GPU for layer operation --- //    
    hipMallocManaged(&d_workspace, workspaceBytes);
    int convout_bytes = convOutDimBatchSize * convOutDimChannels * convOutDimHeight * convOutDimWidth * sizeof(float);    
 
    // memory required for storing output of the conv operation (after adding bias)
    hipMallocManaged(&conv_output, convout_bytes);
    hipMemset(conv_output, 0, convout_bytes);

    // set up memory for pool tensor if pool is true
    if(POOL){
      int poolSize =  outDims.Batch * outDims.Channels * outDims.Height * outDims.Width * sizeof(float);
      hipMallocManaged(&poolTensor, poolSize); 
      hipMemset(poolTensor, 0, poolSize);

    }

    /*
    cerr << "Workspace size: " << (workspaceBytes / 1048576.0) << "MB" << endl;
    */
}


void ConvLayers::fwdProp(){

  checkCUDNN(hipdnnConvolutionForward(CUDNN,
                                      &alph,
                                      input_descriptor,
                                      inputTensor,
                                      kernel_descriptor,
                                      kernelTensor,
                                      convolution_descriptor,
                                      convolution_algorithm,
                                      d_workspace,
                                      workspaceBytes,
                                      &bet,
                                      convOutput_descriptor,
                                      conv_output));

  checkCUDNN(hipdnnAddTensor(CUDNN, &alph, bias_descriptor,
                                  biasTensor ,&bet, convOutput_descriptor, conv_output));

  checkCUDNN(hipdnnActivationForward(CUDNN,
                                      activation_descriptor,
                                      &alph,
                                      convOutput_descriptor,
                                      conv_output,
                                      &bet,
                                      convOutput_descriptor,
                                      conv_output));

  if(POOL){

    checkCUDNN(hipdnnPoolingForward(CUDNN,
                                  pooling_descriptor,
                                  &alph,
                                  convOutput_descriptor,
                                  conv_output,
                                  &bet,
                                  poolTensor_descriptor,
                                  poolTensor));

    hipMallocManaged(&outputTensor,sizeof(poolTensor));
    hipMemcpy(outputTensor,poolTensor,sizeof(poolTensor),hipMemcpyDeviceToDevice);

  }
  else{
	
    hipMallocManaged(&outputTensor,sizeof(conv_output));
    hipMemcpy(outputTensor,conv_output,sizeof(conv_output),hipMemcpyDeviceToDevice);  	
  }


}


// __global__
// void MSE(int num_outputs, float* pred, float* labels, float* cost){


// 	int idx = blockIdx.x * blockDim.x + threadIdx.x;
// 	// int idy = blockIdx.y * blockDim.y + threadIdx.y;


// 	if(idx < num_outputs && idy < batch){

// 		cost[idx] += (pred[idx] - labels[idx]) * (pred[idx] - labels[idx]) / num_outputs;

// 	}

// }

// __global__
// void prepMSE(int outDims, int batch, float *pred, float *labels, float* cost){

// 	int idx = blockIdx.x * blockDim.x + threadIdx.x;
// 	int idy = blockIdx.y * blockDim.y + threadIdx.y;

	
// 	for(int i = idy; i < batch; i++){
// 		for(int j = 0; j < outDims; j++){

// 			cost[i] += (pred[i * outDims + j] - labels[j]);
// 			cost[i] *= cost[i] / outDims;
// 		}
// 	}

// }

// this is del_MSE / del_output = [outDims X batch] matrix
// but in this applicaton, data is real time, bacth size is 1

__global__
void dMSE(int outDims, int batch, float* pred, float* labels, float *dcost){


	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	// int idy = blockIdx.y * blockDim.y + threadIdx.y;

	while(idx < outDims * batch){

		dcost[0] += (2 / outDims) * pred[idx];  

	}

	idx = blockIdx.x * blockDim.x + threadIdx.x;

	while(idx < outDims){

		dcost[0] -= 2 * (batch / outDims) * labels[idx];   

	}


	// int idx = blockIdx.x * blockDim.x + threadIdx.s;

	// while(idx < outDims * batch){

	// 	dcost[idx] = 0;

	// 	for(int i = idx; i < outDims * batch; i += outDims){

	// 		dcost[idx] += (2/outDims) * (  )

	// 	}

	// }
	
}

__global__
void dReLU(int dims, int batch, float* activations, float* dReLU){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < dims * batch){

		if(activations[idx] > 0) dReLU[idx] = 1;
		else dReLU[idx] = 0;

	}

}





// account for the batch size here 



// __global__
// void MSEBackProp(int num_outputs, float* batch, float* pred, float* labels, float* del){

// 	int idx = blockIdx.x * blockDim.x + threadIdx.x;


// 	if(idx < num_outputs){
// 		del[idx] += (2/num_outputs) * (pred[idx] - labels[idx]);
// 	}

// }




void ConvLayers::bwdProp(){


}


__global__
void addBiasFC(int dim1,int dim2, float* bias, float* res){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < dim1*dim2){
		res[idx] += bias[idx];
	}
}


class FCLayers : public ConvLayers{

  public:

    hipblasHandle_t CUBLAS;
    int inDims;
    int outDims;
    int batch;
    float *weights{nullptr};
    float *dcost{nullptr};
    float *labels{nullptr};
    // float *p_act{nullptr};
    float *nabla_w{nullptr};
    float *nabla_b{nullptr};
    float* ones{nullptr};
    float* d_intermediate{nullptr};
    float* dReLU_tensor{nullptr};
		


    bool last;
    // float* outputTensor{nullptr};
    hipdnnTensorDescriptor_t outputTensor_descriptor;

    FCLayers( float* inputTensor_, int inDims_, int batch,int outDims_, float alpha, float beta, hipdnnActivationMode_t ActivationMode_, 
    	hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type,hipblasHandle_t CUBLAS_,hipdnnHandle_t CUDNN){

      this->last = false;
      this->inputTensor = inputTensor_;
      this->inDims = inDims_;
      this->batch = batch;
      this->outDims = outDims_;
      this->CUBLAS = CUBLAS_;
      this->ActivationMode = ActivationMode_;      
      this->alph = alpha;
      this->bet = beta;
      this->CUDNN = CUDNN;
      this->DataType = d_type;
      this->TensorFormat = t_format;
    }

    FCLayers( float* inputTensor_, int inDims_, int batch,int outDims_, float alpha, float beta, hipdnnActivationMode_t ActivationMode_, 
    	hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type,hipblasHandle_t CUBLAS_,hipdnnHandle_t CUDNN, float* labels){

      this->last = true;
      this->inputTensor = inputTensor_;
      this->inDims = inDims_;
      this->batch = batch;
      this->outDims = outDims_;
      this->CUBLAS = CUBLAS_;
      this->ActivationMode = ActivationMode_;      
      this->alph = alpha;
      this->bet = beta;
      this->CUDNN = CUDNN;
      this->DataType = d_type;
      this->TensorFormat = t_format;
    }

    void getFCLayerSpecs();
    void buildFCLayer();
    void fwdProp();
    void bwdProp();

	private:
		int numBlocks;
		int numThreads;


};


void FCLayers::getFCLayerSpecs(){

	checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
	checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
		                                ActivationMode,
		                                HIPDNN_PROPAGATE_NAN,
	    	                            /*relu_coef=*/0));

	checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor_descriptor));
  	checkCUDNN(hipdnnSetTensor4dDescriptor(outputTensor_descriptor,
                                      TensorFormat,
                                      DataType,
                                      batch, outDims, 1, 1));


}

void FCLayers::buildFCLayer(){

	// Initialization of weight matrix

	//Callibrator for weight initialization
	float callibrator = sqrt(2/inDims);

	float *hweights;
	hweights = (float*)malloc(sizeof(float)*inDims*outDims);
	for(int i = 0; i < outDims; i++){
	 hweights[i*outDims + inDims] = d(gen)*callibrator;
	}

	hipMallocManaged(&weights, inDims*outDims*sizeof(float));
	hipMemcpy(weights, hweights, inDims * outDims * sizeof(float),hipMemcpyHostToDevice);

	free(hweights);
	// initialization of bias vector
	hipMallocManaged(&biasTensor,outDims*batch*sizeof(float));
	hipMemset(biasTensor,BIAS_INIT_VAL,outDims*batch*sizeof(float));

	hipMallocManaged(&outputTensor,outDims*batch*sizeof(float));
	// hipMemset(outputTensor,0,outDims*batch*sizeof(float)); ----- If not using hipMemset(),
	// ensure that while performing any operation on it, its multiplicatio coeff is 0, like in hipblasSgemm() below, bet is 0 for the same reason

	// Decide the number of threads and blocks based on the size of the output of the FC layer (before adding the bias) for addBiasFC kernel
	if(batch*outDims <= MAX_THREADS_PER_BLOCK){
		numThreads = batch*outDims;
		numBlocks = 1;
	}
	else{
		numBlocks = roundUp(batch*outDims,MAX_THREADS_PER_BLOCK);
		numThreads = MAX_THREADS_PER_BLOCK;
	}

	// for back prop 
	hipMallocManaged(&ones,sizeof(float) * inDims);
	hipMemset(ones,1,sizeof(ones));
	
	hipMallocManaged(&d_intermediate,sizeof(float) * batch);

	hipMallocManaged(&nabla_w, sizeof(float) * inDims * batch);

	hipMallocManaged(&nabla_b, sizeof(float) * inDims * batch);	

	hipMallocManaged(&dReLU_tensor,sizeof(float) * outDims * batch);
}


void FCLayers::fwdProp(){

	// do w'X + b = weights' * input from previous layer + bias
	// weights = inDims x outDims 
	// bias = outDims x batch
	// output = outDims x batch

	hipblasSgemm(CUBLAS,
				HIPBLAS_OP_T,HIPBLAS_OP_N,
				outDims, batch, inDims,
				&alph,
				weights,inDims,
				inputTensor,inDims,
				&bet,
				outputTensor,outDims);
	
	addBiasFC<<<numBlocks,numThreads>>>(outDims,batch,biasTensor,outputTensor);
	
	checkCUDNN(hipdnnActivationForward(CUDNN,
									activation_descriptor,
									&alph,
									outputTensor_descriptor,
									outputTensor,
									&bet,
									outputTensor_descriptor,
									outputTensor));
	

}


void FCLayers::bwdProp(){

	if(last == true){

		// find the derivative of the cost function
		hipMallocManaged(&dcost,sizeof(float)*outDims);
		hipMemset(dcost,0,sizeof(float)*outDims);
		hipMallocManaged(&labels,sizeof(float)*outDims);

		this->labels = labels;

		if(outDims * batch <= MAX_THREADS_PER_BLOCK){
			numThreads = outDims;
			numBlocks = 1;
		}
		else{

			numBlocks = roundUp(outDims*batch,MAX_THREADS_PER_BLOCK);
			numThreads = MAX_THREADS_PER_BLOCK;
		}

		// calculate del_MSE / del_output (batched)

		dMSE<<<numBlocks,numThreads>>>(outDims, BATCH_SIZE, outputTensor, labels, dcost);

		// calculate gradient of cost with respect to weights

		// del_MSE/del_w_last_layer = ddMSE * del_activation_last_layer/del_last_layer * del_last/del_w_last_layer 


		// calculate del_cost / del_activation = dcost * del_ReLU(z) / del(z) = dcost * dReLU =  [outDims X batchsize]' * [outDims X 1]


		// compute derivative of ReLU and store in dReLU_tensor
		dReLU<<<numBlocks,numThreads>>>(outDims, batch, outputTensor, dReLU_tensor);

		hipblasSgemm(CUBLAS,
				HIPBLAS_OP_T,HIPBLAS_OP_N,
				batch, 1, outDims,
				&alph,
				dReLU_tensor,outDims,
				dcost,outDims,
				&bet,
				d_intermediate,batch);	

		// d_intermediate is (del_MSE / del_output) * (del_activation / del_input) = [batch X 1]

		// now, del_input / del_weights = output of previous layer = input to this layer, which is already there. Therefore, del_MSE / del_w can be given as d_intermediate * inputTensor
		// [inDims X 1] * [batch X 1]' = [inDims X batch]

		hipblasSgemm(CUBLAS,
				HIPBLAS_OP_N,HIPBLAS_OP_T,
				inDims, batch, 1,
				&alph,
				inputTensor, inDims,
				d_intermediate, batch,
				&bet,
				nabla_w, inDims); 

		// calculate gradient of cost with respect to bais

		// d_intermediate can be used to calculate nabla_b as well
		// nabla_b = d_intermediate * [1 1 1 1.... ]' = [batch X 1] * [inDims X 1]' = [batch X inDims]

		hipblasSgemm(CUBLAS,
				HIPBLAS_OP_N,HIPBLAS_OP_T,
				batch, inDims, 1,
				&alph,
				d_intermediate, batch,
				ones, inDims,
				&bet,
				nabla_b, batch);

	}

	else{



	}

	


}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


// __global__
// void trial(int *a, int *b, int M, int N){

// 	int idx = blockIdx.x * blockDim.x + threadIdx.x;
// 	int idy = blockIdx.x * blockDim.x + threadIdx.x;
// 	if(idx < M && idy < N){

// 		b[idx] +=  a[idx + M * idy];
// 		// printf("%d %d\n",idx,idy);

// 	}


// }



int main(int argc, const char* argv[]){

	if (argc < 2) {
	    cerr << "usage: conv <image> [gpu=0] [sigmoid=0]" << endl;
	    exit(EXIT_FAILURE);
  	}

  	gpu_id = (argc > 2) ? std::atoi(argv[2]) : 0;
  	std::cerr << "GPU: " << gpu_id << std::endl;

	hipSetDevice(gpu_id);

	Mat image = load_image(argv[1]);

  	float *inputImage = image2array(image);
	
	//--- Build the Handle for the present layer ---//
	//--- Common for one GPU Device, and all layers of CNN built on it ---//
	hipdnnHandle_t cudnn;
	checkCUDNN(hipdnnCreate(&cudnn));
	hipblasHandle_t cublas;
	hipblasCreate(&cublas);


	convDim_t firstLayerInputDims; //dimensions of input to first layer
	//The input layer will be set here but will be given in each epoch. shift this for loop
	firstLayerInputDims.Height = image.rows;
	firstLayerInputDims.Width = image.cols;
	firstLayerInputDims.Channels = image.channels();
	firstLayerInputDims.Batch = BATCH_SIZE;
  
	float *input_layer1;
	hipMallocManaged(&input_layer1,firstLayerInputDims.Height * firstLayerInputDims.Width * firstLayerInputDims.Channels * firstLayerInputDims.Batch * sizeof(float));
	hipMemcpy(input_layer1,inputImage,sizeof(inputImage),hipMemcpyHostToDevice);


	// start with the kernel specs, according to the following
	// kernelDim_t setKernelSpecs(int size, int fheight, int fwidth, int sheight, int swidth, int pheight, int pwidth, int dheight, int dwidth)

	kernelDim_t layerKernel1 = setKernelSpecs(3,5,5,2,2,1,1,1,1);

	///////////////////////// set pooling specs like this, if there is a pooling layer after your conv layer/////////////////////////////////////////
	// poolDim_t poolDim1 = setPoolSpecs((bool)OVERLAP_POOLING); //setting a overlapping pool layer
	////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

	/*
	This is the signature for conv layer without pooling layer after it.

	ConvLayers( int index, float* inT, convDim_t inDim, kernelDim_t kdims, int a, int b,
	hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type, hipdnnConvolutionMode_t c_mode, hipdnnActivationMode_t ActMode,hipdnnPoolingMode_t poolMode, hipdnnHandle_t cud);

	This is the signature for conv layer with pooling layer after it

	ConvLayers( int index, float* inT, convDim_t inDim, kernelDim_t kdims, poolDim_t pdims, int a, int b,
	hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type, hipdnnConvolutionMode_t c_mode, hipdnnActivationMode_t ActMode,hipdnnPoolingMode_t poolMode, hipdnnHandle_t cud); 
	*/

	// Make the architecture

	// This is the convolutional layer constructor (without pooling layer)
	ConvLayers convlayer1(1, input_layer1, firstLayerInputDims, layerKernel1, alpha, beta, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, HIPDNN_CROSS_CORRELATION,
	                    HIPDNN_ACTIVATION_RELU, cudnn);
	// with these functions, we allocate the space in GPU for layer operation. 
	convlayer1.getConvLayerSpecs();
	convlayer1.buildConvLayer(); // this is objectName.conv_output. If you have added a pool layer, use objectName.poolTensor as output of this layer for input of next layer 
	                           // Here, memory is allocated and values is initialized to zero but no computation has been done yet.
	convlayer1.fwdProp();

	// At this point, we have defined the layer, but we havent implemented forward or backward prop. That will be done while we start training, while this is just defination
	// memory allocation for forward pass (backward pass to be implemented after ths)

	// create another layer, this time with pool, therefore the signature of conv constructor will be different from the previous layer (Declare poolDim_t
	// if you want pooling layer)
	poolDim_t poolDim2 = setPoolSpecs(!(bool)OVERLAP_POOLING);

	kernelDim_t layerKernel2 = setKernelSpecs(3,5,5,2,2,1,1,1,1);
	ConvLayers convlayer2(2, convlayer1.conv_output, convlayer1.outDims, layerKernel2, poolDim2, alpha, beta, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, HIPDNN_CROSS_CORRELATION,
	                    HIPDNN_ACTIVATION_RELU, HIPDNN_POOLING_MAX, cudnn);
	convlayer2.getConvLayerSpecs();
	convlayer2.buildConvLayer();
	convlayer2.fwdProp();



	// FCLayers( float* inputTensor_, int inDims_, int batch,int outDims_, float alpha, float beta, hipdnnActivationMode_t ActivationMode_, 
 	//    	hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type,hipblasHandle_t CUBLAS_,hipdnnHandle_t CUDNN)

	// build a fully connected layer
	int fclayer1_input_dims = convlayer2.outDims.Height * convlayer2.outDims.Width * convlayer2.outDims.Channels; 

	FCLayers fclayer1( convlayer2.outputTensor, fclayer1_input_dims, BATCH_SIZE, 100 ,1.0, 0.0, HIPDNN_ACTIVATION_RELU,HIPDNN_TENSOR_NHWC,
						HIPDNN_DATA_FLOAT, cublas,cudnn);
	fclayer1.getFCLayerSpecs();
	fclayer1.buildFCLayer();
	fclayer1.fwdProp();

	// dummy labels
	float* labels;
	hipMallocManaged(&labels, sizeof(float));
	hipMemset(labels,1.0,sizeof(float));

	FCLayers fclayer2(fclayer1.outputTensor, fclayer1.outDims, BATCH_SIZE, 1 ,1.0, 0.0, HIPDNN_ACTIVATION_RELU,HIPDNN_TENSOR_NHWC,
						HIPDNN_DATA_FLOAT, cublas,cudnn, labels);
	fclayer2.getFCLayerSpecs();
	fclayer2.buildFCLayer();
	fclayer2.fwdProp();

	

	float *output_image{nullptr};
	output_image = (float*)malloc(sizeof(convlayer2.outputTensor));
	hipMemcpy(output_image, convlayer2.outputTensor, sizeof(convlayer2.outputTensor),hipMemcpyDeviceToHost);

	hipDeviceSynchronize();




	// for(int i = 0; i < 100*1; i++){
	// 	cout<<output_image[i]<<endl;
	// }
	// int dim1 = 10,dim2 = 10;

	// Mat m = Mat(convlayer2.outDims.Width,convlayer2.outDims.Height,CV_8UC1);
	// memcpy(m.data,output_image,sizeof(output_image));


	string output_filename = "/generated_images";
	// save_image(output_filename, output_image, convlayer2.outDims.Width, convlayer2.outDims.Height);
	// imwrite(output_filename, m);
 	cerr << "Wrote output to " << output_filename << std::endl;


	cout<<CUDNN_MAJOR;
	

	return 0;
}
