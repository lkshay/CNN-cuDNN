#include "hip/hip_runtime.h"
#include <hipDNN.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <opencv2/opencv.hpp>
#include <string>
#include <random>
#include <cmath>
using namespace std;
using namespace cv;
#define BATCH_SIZE 1
#define OVERLAP_POOLING 1
#define BIAS_INIT_VAL 0.001

int gpu_id;
int device;

int roundUp(int num, int den){

  return((num + den - 1 )/(den));

}

// a struct for outputs & inputs of convolutional layers (including max pool, if exists)

struct convDim_t{

  int Height;
  int Width;
  int Channels;
  int Batch;
};

// a struct to define pooling layers for a conv layer (if pool is true)
struct poolDim_t{

  int Height;
  int Width;
  int padHeight;
  int padWidth;
  int strideHeight;
  int strideWidth;
};

// a struct to define kernel dimensions for a conv operation in a layer

struct kernelDim_t{

  int kernelSize;
  int kernelHeight;
  int kernelWidth;
  int strideHeight;
  int strideWidth;
  int padHeight;
  int padWidth;
  int dilationHeight;
  int dilationWidth;
};


// a function to set the kernel params for a conv operation in a lyer

kernelDim_t setKernelSpecs(int size, int fheight, int fwidth, int sheight, int swidth, int pheight, int pwidth, int dheight, int dwidth){

  kernelDim_t layerKernel;
  layerKernel.kernelSize = size;
  layerKernel.kernelHeight = fheight;
  layerKernel.kernelWidth = fwidth;
  layerKernel.strideHeight = sheight;
  layerKernel.strideWidth = swidth;
  layerKernel.padHeight = pheight;
  layerKernel.padWidth = pwidth;
  layerKernel.dilationHeight = dheight;
  layerKernel.dilationWidth = dwidth;

  return layerKernel;
}

/*
int flagOverlap is a flag for setting dimensions ov poolDims. If it is 1, then F=3,S=2 otherwise F=2,S=2.
It is worth noting that there are only two commonly seen variations of the max pooling layer found in practice: 
A pooling layer with F=3,S=2 (also called overlapping pooling), and more commonly F=2,S=2. Pooling sizes with larger receptive fields are too destructive.
*/

// a function to set pool dimensions for a layer operation, if pool is true

poolDim_t setPoolSpecs(bool flagOverlap){

  poolDim_t poolDims;

  if(flagOverlap){

    poolDims.Height = 3;
    poolDims.Width = 3;
    poolDims.padHeight = 1;
    poolDims.padWidth = 1;
    poolDims.strideHeight = 2;
    poolDims.strideWidth = 2;  
  }
  else{
    poolDims.Height = 2;
    poolDims.Width = 2;
    poolDims.padHeight = 1;
    poolDims.padWidth = 1;
    poolDims.strideHeight = 2;
    poolDims.strideWidth = 2;
  }

  return poolDims;
  
}


#define checkCUDNN(expression)                             \
{                                                          \
  hipdnnStatus_t status = (expression);                     \
  if (status != HIPDNN_STATUS_SUCCESS) {                    \
    std::cerr << "Error on line " << __LINE__ << ": "      \
              << hipdnnGetErrorString(status) << std::endl; \
    std::exit(EXIT_FAILURE);                               \
  }                                                        \
}


// --- A function to convert the image to a array to be passed into the input conv layer --- //

float * image2array(Mat image){

  float *imageArray = (float *)image.data;
  
  return imageArray;
}

void save_image(const string output_filename,
                float* buffer,
                int height,
                int width) {
  cv::Mat output_image(height, width, CV_32FC3, buffer);
  // Make negative values zero.
  cv::threshold(output_image,
                output_image,
                /*threshold=*/0,
                /*maxval=*/0,
                cv::THRESH_TOZERO);
  cv::normalize(output_image, output_image, 0.0, 255.0, cv::NORM_MINMAX);
  output_image.convertTo(output_image, CV_8UC3);
  cv::imwrite(output_filename, output_image);
  std::cerr << "Wrote output to " << output_filename << std::endl;
}

Mat load_image(const char* image_path) {
  cv::Mat image = cv::imread(image_path, CV_LOAD_IMAGE_COLOR);
  image.convertTo(image, CV_32FC3);
  cv::normalize(image, image, 0, 1, cv::NORM_MINMAX);
  cerr << "Input Image: " << image.rows << " x " << image.cols << " x "
          << image.channels() << endl;
  return image;
}

/*

-Prepare the kernel and bias tensors before sending them to the next layer. Initialize the kernel with He. et. al., and bias with zero.
-The filter initializer is common to all the layers, so is the bias. The channel layout is also same except in some cases (will find out later)
-Input to the layer is now the dimension of kernel tensor, bias tensor, input tensor, and the initialized bias and kernel tensors and the
 and the input tensor either from a previous layer or the input image

*/

float alpha = 1.0;
float beta = 0.0;

class ConvLayers{

	// In addition to the inHeight, inWidth and inChannels pertaining to the output of the previous layer,
	// also have output object of the previous layer as a member of this class.
  public:

  float *inputTensor;                   // pointer to the input tensor. If this is an input layer, convert the cv::Mat image to a 3-D array first and then pass its pointer 
                                        // to the class constructor 
  	float *kernelTensor;
  	float *biasTensor;		
	int layerIndex;
  	int alph, bet;
  	hipdnnHandle_t CUDNN;
  	hipdnnTensorFormat_t TensorFormat;
	hipdnnDataType_t DataType;
	hipdnnConvolutionMode_t ConvMode;
  	hipdnnActivationMode_t ActivationMode;
	hipdnnPoolingMode_t PoolingMode;
    convDim_t outDims;
    convDim_t inDims;
  	kernelDim_t kernelDims;
	poolDim_t poolDims;

  random_device rd{};
  mt19937 gen{rd()};  
  normal_distribution<> d{0,1}; 

  float* conv_output{nullptr}; // output of convolution
  float* poolTensor{nullptr};  // output of pooling layer, if exists
  void* d_workspace{nullptr};
  size_t workspaceBytes{0};

  int convOutDimHeight{0}, convOutDimWidth{0}, convOutDimChannels{0}, convOutDimBatchSize{0};
  int poolOutBatchSize{0}, poolOutChannels{0}, poolOutHeight{0}, poolOutWidth{0};

  bool POOL;  // True if pooling is to be done in this layer, otherwise False

  hipdnnTensorDescriptor_t input_descriptor;
  hipdnnFilterDescriptor_t kernel_descriptor;
  hipdnnConvolutionDescriptor_t convolution_descriptor;
  hipdnnTensorDescriptor_t bias_descriptor;
  hipdnnTensorDescriptor_t convOutput_descriptor;
  hipdnnConvolutionFwdAlgo_t convolution_algorithm;
  hipdnnActivationDescriptor_t activation_descriptor;
  hipdnnPoolingDescriptor_t pooling_descriptor;
  hipdnnTensorDescriptor_t poolTensor_descriptor;
	  
  /*
  Constructor overloading for initialiing the class object with or without pooling mode. If the user wants to use pooling layer, use the second signature, otherwise first.
  The ConvLayers class' objects behave differently when a pool layer is to be used and differently when pool isnt there!
  */

	ConvLayers( int index, float* inT, convDim_t inDim, kernelDim_t kdims, int a, int b, 
		hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type, hipdnnConvolutionMode_t c_mode, hipdnnActivationMode_t ActMode, hipdnnHandle_t cud){

    this->POOL = false;
    this->inputTensor = inT;
    this->inDims = inDim;
    this->kernelDims = kdims;
    
    this->layerIndex = index;
    this->alph = a; this->bet = b;
	this->TensorFormat = t_format;
	this->DataType = d_type;			
	this->ConvMode = c_mode;
    this->ActivationMode = ActMode;
	this->CUDNN = cud;	
	}

  ConvLayers( int index, float* inT, convDim_t inDim, kernelDim_t kdims, poolDim_t pdims, int a, int b, 
    hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type, hipdnnConvolutionMode_t c_mode, hipdnnActivationMode_t ActMode,hipdnnPoolingMode_t poolMode, hipdnnHandle_t cud){

    this->POOL = true;
    this->inputTensor = inT;
    this->inDims = inDim;
    this->kernelDims = kdims;
    this->poolDims = pdims;
    this->layerIndex = index;
    this->alph = a; this->bet = b;
    this->TensorFormat = t_format;
    this->DataType = d_type;      
    this->ConvMode = c_mode;
    this->ActivationMode = ActMode;
    this->PoolingMode = poolMode;
    this->CUDNN = cud;  
  }

  	void getConvLayerSpecs();

	void buildConvLayer();

  	void fwdProp();

  	void bwdPropo();

};

  void ConvLayers::getConvLayerSpecs(){



  checkCUDNN(hipdnnCreateTensorDescriptor(&input_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(input_descriptor,
                                          TensorFormat,
                                          DataType,
                                          inDims.Batch,
                                          inDims.Channels, 
                                          inDims.Height, 
                                          inDims.Width));

   
  // --- Build the Kernel which is going to convolve over the input ---//
  
  
  checkCUDNN(hipdnnCreateFilterDescriptor(&kernel_descriptor));
  checkCUDNN(hipdnnSetFilter4dDescriptor(kernel_descriptor,
                                        DataType,
                                        TensorFormat,
                                        kernelDims.kernelSize,
                                        inDims.Channels,
                                        kernelDims.kernelHeight,
                                        kernelDims.kernelWidth));

  // --- Build the Convolution descriptor --- //

  
  checkCUDNN(hipdnnCreateConvolutionDescriptor(&convolution_descriptor));
  checkCUDNN(hipdnnSetConvolution2dDescriptor(convolution_descriptor,
                                            kernelDims.padHeight,
                                            kernelDims.padWidth,
                                            kernelDims.strideHeight,
                                            kernelDims.strideWidth,
                                            kernelDims.dilationHeight,
                                            kernelDims.dilationWidth,
                                            ConvMode,
                                            DataType));

  // --- This function returns the dimensions of the resulting 4D tensor of a 2D convolution,     //
  // ---given the convolution descriptor, the input tensor descriptor and the filter descriptor --- //

  checkCUDNN(hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor,
                                                 input_descriptor,
                                                 kernel_descriptor,
                                                 &convOutDimBatchSize,
                                                 &convOutDimChannels,
                                                 &convOutDimHeight,
                                                 &convOutDimWidth));
  
  outDims.Height = convOutDimHeight;
  outDims.Width = convOutDimWidth;
  outDims.Channels = convOutDimChannels;
  outDims.Batch = convOutDimBatchSize;
  
  cout<<"Output image size "<<outDims.Batch<<" X "<<outDims.Height<<" X "<<outDims.Width<<" X "<<outDims.Channels<<endl;
  
  checkCUDNN(hipdnnCreateTensorDescriptor(&bias_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(bias_descriptor,
                                            TensorFormat,
                                            DataType,
                                            convOutDimBatchSize,
                                           convOutDimChannels,
                                           convOutDimHeight,
                                           convOutDimWidth));

  // ---Build the output Descriptor ---//

  
  checkCUDNN(hipdnnCreateTensorDescriptor(&convOutput_descriptor));
  checkCUDNN(hipdnnSetTensor4dDescriptor(convOutput_descriptor,
                                        TensorFormat,
                                        DataType, 
                                        convOutDimBatchSize,
                                        convOutDimChannels,
                                        convOutDimHeight,
                                        convOutDimWidth));

  // -- Size references for next conv layer --- //

  

  // --- Determine the Convolution algorithm to be used in CNN layer ---//

  
  checkCUDNN(hipdnnGetConvolutionForwardAlgorithm(CUDNN,
                                        input_descriptor,
                                        kernel_descriptor,
                                        convolution_descriptor,
                                        convOutput_descriptor,
                                        HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                        /*memoryLimitInBytes=*/0,
                                        &convolution_algorithm));

  
  checkCUDNN(hipdnnCreateActivationDescriptor(&activation_descriptor));
  checkCUDNN(hipdnnSetActivationDescriptor(activation_descriptor,
                                        ActivationMode,
                                        HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/0));


  /*
  Do some adjustment if the output dimension of pooling layer is not an integer (which will give an error) 
  Each dimension h and w of the output images is computed as followed:
  outputDim = 1 + (inputDim + 2*padding - windowDim)/poolingStride;

  */

  // check if the user has asked to create a pooling layer for this conv layer
  if(POOL){

    if((outDims.Height - poolDims.Height)%2 != 0){
      poolDims.Height = (poolDims.Height == 2) ? 3 : 2;
    }

    if((outDims.Width - poolDims.Width)%2 != 0){
      poolDims.Width = (poolDims.Width == 2) ? 3 : 2;
    }

    
    checkCUDNN(hipdnnCreatePoolingDescriptor(&pooling_descriptor));
    checkCUDNN(hipdnnSetPooling2dDescriptor(pooling_descriptor,
                                            PoolingMode,
                                            HIPDNN_NOT_PROPAGATE_NAN,
                                            poolDims.Height,
                                            poolDims.Width,
                                            poolDims.padHeight,
                                            poolDims.padWidth,
                                            poolDims.strideHeight,
                                            poolDims.strideWidth));

    checkCUDNN(hipdnnGetPooling2dForwardOutputDim(pooling_descriptor,
                                              convOutput_descriptor,
                                                  &poolOutBatchSize,
                                                  &poolOutChannels,
                                                  &poolOutHeight,
                                                  &poolOutWidth));

    
    checkCUDNN(hipdnnCreateTensorDescriptor(&poolTensor_descriptor));  
    checkCUDNN(hipdnnSetTensor4dDescriptor(poolTensor_descriptor,
                                          TensorFormat,
                                          DataType,
                                          poolOutBatchSize,
                                          poolOutChannels,
                                          poolOutHeight,
                                          poolOutWidth));

    outDims.Batch = poolOutBatchSize;
    outDims.Channels = poolOutChannels;
    outDims.Height = poolOutHeight;
    outDims.Width = poolOutWidth;

    }

    

  }

  void ConvLayers::buildConvLayer(){

  	// --- Set up the memory required for the convolution --- //
  
	  
    checkCUDNN(hipdnnGetConvolutionForwardWorkspaceSize(CUDNN,
                                                     input_descriptor,
                                                     kernel_descriptor,
                                                     convolution_descriptor,
                                                     convOutput_descriptor,
                                                     convolution_algorithm,
                                                     &workspaceBytes));

    // Initialize bias and kernel tensors here //

    // Bias
    hipMallocManaged(&biasTensor, outDims.Channels * outDims.Batch * sizeof(float));
    hipMemset(biasTensor,(float)BIAS_INIT_VAL,outDims.Channels * outDims.Batch * sizeof(float)); //initializing all the bias units to BIAS_INIT_VAL

    //Kernel
    //random_device rd{};
    //mt19937 gen{rd()};

    // for initialization of weights with gassian distribution with zero mean and variance as one
    //normal_distribution<> d{0,1}; 

    // callibrator to be multplied with the weights for scaling according to He. et. al. 
    float callibrator = (layerIndex != 1) ? sqrt(2 / (inDims.Channels * inDims.Height * inDims.Width)) : 1.0;

    float kernelTemplate[kernelDims.kernelHeight][kernelDims.kernelWidth];
    for(int i = 0; i < kernelDims.kernelHeight; i++){
      for(int j = 0; j < kernelDims.kernelWidth; j++){
        kernelTemplate[i][j] = d(gen) * callibrator;
      }
    }
    float hkernel[kernelDims.kernelSize][inDims.Channels][kernelDims.kernelHeight][kernelDims.kernelWidth];

    for(int i = 0; i < kernelDims.kernelSize; i++){
      for(int j = 0; j < inDims.Channels; j++){
        for(int k = 0; k < kernelDims.kernelHeight; k++){
          for(int l = 0; l < kernelDims.kernelWidth; l++){
            hkernel[i][j][k][l] = kernelTemplate[k][l]; 
          }
        }
      }
    }

    hipMallocManaged(&kernelTensor, kernelDims.kernelSize * kernelDims.kernelHeight * kernelDims.kernelWidth * sizeof(float));
    hipMemcpy(kernelTensor,hkernel,sizeof(hkernel),hipMemcpyHostToDevice);
    
    // --- Allocate Memory in the GPU for layer operation --- //    
    hipMallocManaged(&d_workspace, workspaceBytes);
    int convout_bytes = convOutDimBatchSize * convOutDimChannels * convOutDimHeight * convOutDimWidth * sizeof(float);    
 
    // memory required for storing output of the conv operation (after adding bias)
    hipMallocManaged(&conv_output, convout_bytes);
    hipMemset(conv_output, 0, convout_bytes);

    // set up memory for pool tensor if pool is true
    if(POOL){
      int poolSize =  outDims.Batch * outDims.Channels * outDims.Height * outDims.Width * sizeof(float);
      hipMallocManaged(&poolTensor, poolSize); 
      hipMemset(poolTensor, 0, poolSize);

    }

    /*
    cerr << "Workspace size: " << (workspaceBytes / 1048576.0) << "MB" << endl;
    */
}


void ConvLayers::fwdProp(){

  checkCUDNN(hipdnnConvolutionForward(CUDNN,
                                      &alph,
                                      input_descriptor,
                                      inputTensor,
                                      kernel_descriptor,
                                      kernelTensor,
                                      convolution_descriptor,
                                      convolution_algorithm,
                                      d_workspace,
                                      workspaceBytes,
                                      &bet,
                                      convOutput_descriptor,
                                      conv_output));

  checkCUDNN(hipdnnAddTensor(CUDNN, &alph, bias_descriptor,
                                  biasTensor ,&bet, convOutput_descriptor, conv_output));

  checkCUDNN(hipdnnActivationForward(CUDNN,
                                      activation_descriptor,
                                      &alph,
                                      convOutput_descriptor,
                                      conv_output,
                                      &bet,
                                      convOutput_descriptor,
                                      conv_output));

  if(POOL){

    checkCUDNN(hipdnnPoolingForward(CUDNN,
                                  pooling_descriptor,
                                  &alph,
                                  convOutput_descriptor,
                                  conv_output,
                                  &bet,
                                  poolTensor_descriptor,
                                  poolTensor));
  }


}





/*
void ConvLayers::fwdProp(){

  checkCUDNN(hipdnnConvolutionForward(CUDNN,
                                     &alph,
                                     layerSpecs.input_desc,
                                     inputTensor,
                                     layerSpecs.kernel_desc,
                                     kernelTensor,
                                     layerSpecs.convolution_desc,
                                     layerSpecs.convolution_algo,
                                     d_workspace,
                                     workspaceBytes,
                                     &bet,
                                     layerSpecs.output_desc,
                                     conv_output));

  checkCUDNN(hipdnnActivationForward(CUDNN,
                                    layerSpecs.activation_desc,
                                    &alph,
                                    layerSpecs.output_desc,
                                    conv_output,
                                    &bet,
                                    layerSpecs.output_desc,
                                    conv_output));

  checkCUDNN(hipdnnAddTensor(CUDNN, &alph, layerSpecs.bias_desc,
                                  biasTensor ,&bet, layerSpecs.output_desc, conv_output));
  
  if(POOL){
    checkCUDNN(hipdnnPoolingForward(CUDNN, layerSpecs.pooling_desc, &alph, layerSpecs.output_desc,
                                       conv_output, &bet, layerSpecs.poolTensor_desc, poolTensor));
  }

}
*/

class FCLayers{

  public:
    hipblasHandle_t CUBLAS;
    int inDims;
    float* inputTensor;
    float* outputTensor;
    int outDims;
    hipdnnActivationMode_t ActivationMode;
    hipdnnActivationDescriptor_t activationDesc;
    float *weights;
    float *bias;

    // Random nunmber generator for weights
    random_device rdd{};
    mt19937 generator{rdd()};
    normal_distribution<> dist{0,1};


    FCLayers( float* inputTensor_, int inDims_, int outDims_, hipdnnActivationMode_t ActivationMode_, hipblasHandle_t CUBLAS_){

      inputTensor = inputTensor_;
      inDims = inDims_;
      outDims = outDims_;
      CUBLAS = CUBLAS_;
      ActivationMode = ActivationMode_;

    }

    void getFCLayerSpecs();
    void buildFCLayer();

};


void FCLayers::getFCLayerSpecs(){

  checkCUDNN(hipdnnCreateActivationDescriptor(&activationDesc));
  checkCUDNN(hipdnnSetActivationDescriptor(activationDesc,
                                        ActivationMode,
                                        HIPDNN_PROPAGATE_NAN,
                                        /*relu_coef=*/0));
}

void FCLayers::buildFCLayer(){

  // Initialization of weight matrix

  //Callibrator for weight initialization
  float callibrator = sqrt(2/inDims);


  float *hweights;
  hweights = (float*)malloc(sizeof(float)*inDims*outDims);
  for(int i = 0; i < outDims; i++){
    hweights[i*outDims + inDims] = dist(generator)*callibrator;
  }
 
  hipMallocManaged(&weights, inDims*outDims*sizeof(float));
  hipMemcpy(weights, hweights, inDims * outDims * sizeof(float),hipMemcpyHostToDevice);
  
  // initialization of bias vector
  hipMallocManaged(&bias,outDims*sizeof(float));
  hipMemset(bias,BIAS_INIT_VAL,outDims*sizeof(float));

  hipMallocManaged(&outputTensor,outDims*sizeof(float));
  hipMemset(outputTensor,0,outDims*sizeof(float));
  
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, const char* argv[]){

	if (argc < 2) {
    cerr << "usage: conv <image> [gpu=0] [sigmoid=0]" << endl;
    exit(EXIT_FAILURE);
  }

  gpu_id = (argc > 2) ? std::atoi(argv[2]) : 0;
  std::cerr << "GPU: " << gpu_id << std::endl;

  hipSetDevice(gpu_id);

	Mat image = load_image(argv[1]);

  float *inputImage = image2array(image);
	
	//--- Build the Handle for the present layer ---//
	//--- Common for one GPU Device, and all layers of CNN built on it ---//
	hipdnnHandle_t cudnn;
  checkCUDNN(hipdnnCreate(&cudnn));
  hipblasHandle_t cublas;
  hipblasCreate(&cublas);


  convDim_t firstLayerInputDims; //dimensions of input to first layer
  //The input layer will be set here but will be given in each epoch. shift this for loop
  firstLayerInputDims.Height = image.rows;
  firstLayerInputDims.Width = image.cols;
  firstLayerInputDims.Channels = image.channels();
  firstLayerInputDims.Batch = BATCH_SIZE;
  
  float *input_layer1;
  hipMallocManaged(&input_layer1,firstLayerInputDims.Height * firstLayerInputDims.Width * firstLayerInputDims.Channels * firstLayerInputDims.Batch * sizeof(float));
  hipMemcpy(input_layer1,inputImage,sizeof(inputImage),hipMemcpyHostToDevice);


  // start with the kernel specs
  kernelDim_t layerKernel1 = setKernelSpecs(3,5,5,1,1,1,1,1,1);

  ///////////////////////// set pooling specs like this, if there is a pooling layer after your conv layer/////////////////////////////////////////
  // poolDim_t poolDim1 = setPoolSpecs((bool)OVERLAP_POOLING); //setting a overlapping pool layer
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
  
  /*
  This is the signature for conv layer without pooling layer after it.

    ConvLayers( int index, float* inT, convDim_t inDim, kernelDim_t kdims, int a, int b,
    hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type, hipdnnConvolutionMode_t c_mode, hipdnnActivationMode_t ActMode,hipdnnPoolingMode_t poolMode, hipdnnHandle_t cud);

  This is the signature for conv layer with pooling layer after it

    ConvLayers( int index, float* inT, convDim_t inDim, kernelDim_t kdims, poolDim_t pdims, int a, int b,
    hipdnnTensorFormat_t t_format, hipdnnDataType_t d_type, hipdnnConvolutionMode_t c_mode, hipdnnActivationMode_t ActMode,hipdnnPoolingMode_t poolMode, hipdnnHandle_t cud); 
  */
  
  // Make the architecture

  // This is the convolutional layer constructor (without pooling layer)
  ConvLayers convlayer1(1, input_layer1, firstLayerInputDims, layerKernel1, alpha, beta, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, HIPDNN_CROSS_CORRELATION,
                        HIPDNN_ACTIVATION_RELU, cudnn);
  // with these functions, we allocate the space in GPU for layer operation. 
  convlayer1.getConvLayerSpecs();
  convlayer1.buildConvLayer(); // this is objectName.conv_output. If you have added a pool layer, use objectName.poolTensor as output of this layer for input of next layer 
                               // Here, memory is allocated and values is initialized to zero but no computation has been done yet.

  convlayer1.fwdProp();

  // At this point, we have defined the layer, but we havent implemented forward or backward prop. That will be done while we start training, while this is just defination
  // memory allocation for forward pass (backward pass to be implemented after ths)

  // create another layer, this time with pool, therefore the signature of conv constructor will be different from the previous layer (Declare poolDim_t if you want pooling layer)
  poolDim_t poolDim2 = setPoolSpecs((bool)OVERLAP_POOLING);

  kernelDim_t layerKernel2 = setKernelSpecs(3,6,6,1,1,1,1,1,1);
  ConvLayers convlayer2(2, convlayer1.conv_output, convlayer1.outDims, layerKernel2, poolDim2, alpha, beta, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_FLOAT, HIPDNN_CROSS_CORRELATION,
                        HIPDNN_ACTIVATION_RELU, HIPDNN_POOLING_MAX, cudnn);
  convlayer2.getConvLayerSpecs();
  convlayer2.buildConvLayer();

  // build a fully connected layer
  int fclayer1_input_dims = convlayer2.outDims.Height * convlayer2.outDims.Width * convlayer2.outDims.Channels; 

  FCLayers fclayer1( convlayer2.poolTensor, fclayer1_input_dims, 100 , HIPDNN_ACTIVATION_RELU, cublas);
  fclayer1.getFCLayerSpecs();
  fclayer1.buildFCLayer();

  int fclayer2_input_dims = fclayer1.outDims;

  FCLayers fclayer2(fclayer1.outputTensor, fclayer2_input_dims, 50, HIPDNN_ACTIVATION_RELU, cublas); 
  fclayer2.getFCLayerSpecs();
  fclayer2.buildFCLayer();


  hipDeviceSynchronize();

	return 0;
}